#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <ctime>
#include <cmath>
//#include <QDebug>


#define PI 3.14159265359

using namespace std;

template<class T>
class CudaArray
{
public:
    CudaArray (int n) : dev{nullptr},nBytes{0}
    {
        try{
            nBytes = n * sizeof(T);
            hipMalloc((void**)&dev,nBytes);
        } catch (...){}
    }
    ~CudaArray ()
    {
        try{
            hipFree (dev);
        }
        catch(...) {}
    }
    int getnBytes() const
    {
        return nBytes;
    }
    operator T* () const
    {
        return dev;
    }
    CudaArray& operator = (T* rhs) const
    {
        hipMemcpy (dev,rhs,nBytes,hipMemcpyHostToDevice);
    }
    void copyToCpu(T *cpu) const
    {
        hipMemcpy (cpu,dev,nBytes,hipMemcpyDeviceToHost);
    }
private:
    T *dev;
    int nBytes;
};


__device__ float sinteta;
__device__ float costeta;
__global__ void kernelSetsincos(double teta)
{
    sinteta = sin (teta);
    costeta = cos (teta);
}
// http://habrahabr.ru/post/55461/ (c)
#define BLOCK_DIM 16
__global__ void kernelRotate(unsigned int* inputMatrix, unsigned int* outputMatrix, int width, int height)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    if ((xIndex < width) && (yIndex < height))
    {
        //Линейный индекс элемента строки исходной матрицы
        int inputIdx = xIndex + width * yIndex;
        //Линейный индекс элемента столбца матрицы-результата
        xIndex -= width / 2.0;
        yIndex -= height / 2.0;
        int nx = xIndex * costeta + yIndex * sinteta,
                ny = -xIndex * sinteta + yIndex * costeta;
        nx += width / 2.0;
        ny += height / 2.0;
        if (nx >= 0 && nx < width && ny >= 0 && ny < height){
            int outputIdx = nx  + width * ny;
            outputMatrix[outputIdx] = inputMatrix[inputIdx];
        }
    }
    //    __shared__ float temp[BLOCK_DIM][BLOCK_DIM];

    //    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //    if ((xIndex < width) && (yIndex < height))
    //    {
    //        // Линейный индекс элемента строки исходной матрицы
    //        int idx = yIndex * width + xIndex;

    //        //Копируем элементы исходной матрицы
    //        temp[threadIdx.y][threadIdx.x] = inputMatrix[idx];
    //    }

    //    //Синхронизируем все нити в блоке
    //    __syncthreads();

    //    xIndex = blockIdx.y * blockDim.y + threadIdx.x;
    //    yIndex = blockIdx.x * blockDim.x + threadIdx.y;

    //    if ((xIndex < width) && (yIndex < height))
    //    {
    //        //        xIndex -= width / 2.0;
    //        //        yIndex -= height / 2.0;

    //        //        float nx = xIndex * costeta + yIndex * sinteta,
    //        //                ny = -xIndex * sinteta + yIndex * costeta;

    //        //        nx += width / 2.0;
    //        //        ny += height / 2.0;
    //        //        //x[i] = xi * costeta + yi * sinteta;
    //        //        //y[i] = -xi * sinteta + yi * costeta;

    //        //        if (nx >= 0 && nx < width && ny >= 0 && ny < height){

    //        //            // Линейный индекс элемента строки исходной матрицы
    //        int idx = yIndex * width + xIndex;

    //        //Копируем элементы исходной матрицы
    //        outputMatrix[idx] = temp[threadIdx.x][threadIdx.y];
    //        // }

    //    }
}
extern "C" void algoCudaRotate(unsigned char *input_output,int step,int rows,int cols,double angle)
{
    hipMemset(&sinteta,0,sizeof(float));
    hipMemset(&costeta,0,sizeof(float));
    kernelSetsincos<<<1,1>>> (-angle / 180.0 * PI);
   // std::cerr << "sincos set\n";
    int channels = 3,
            width = cols, //cols * channels,
            height = rows,
            n = width * height,
            bytesn = n * sizeof(int);

   // std::cerr << "step: " << step << " rows: " << rows << " cols: " << cols << '\n';
    unsigned int *data = new unsigned int [n];
    for(int i = 0;i < height; ++i){ // Упаковка bgr в int 4 байта тип
        for(int j = 0;j < width * channels; j += channels){
            unsigned char b = input_output[step * i + j],
                    g = input_output[step * i + j + 1],
                    r = input_output[step * i + j + 2];
            data[i * width + j / channels] = (int(b) << 16) | (int(g) << 8) | int(r);
        }
    }
    //std::cerr << "array read\n";
    CudaArray<unsigned int> inputimg(n),rotatedimg(n);
    inputimg = data;
    //std::cerr << "array copy to device\n";

    dim3 gridSize = dim3(width / BLOCK_DIM, height / BLOCK_DIM, 1);
    dim3 blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
    kernelRotate<<< gridSize, blockSize >>> (inputimg,rotatedimg,width,height);
    //std::cerr << "call kernel\n";

    rotatedimg.copyToCpu(data);
  //  std::fill (input_output,input_output + height * step,0);
   // std::cerr << "returen result to cpu mem\n";
    for(int i = 0;i < height; ++i){ // распаковка
        for(int j = 0;j < width * channels; j += channels){
            unsigned int bgr = data[i * width + j / channels];
            input_output[step * i + j] = (bgr >> 16);
            input_output[step * i + j + 1] = (bgr & (255 << 8)) >> 8;
            input_output[step * i + j + 2] = (bgr & 255);
        }
    }
    hipFree (&sinteta);
    hipFree (&costeta);
    delete [] data;

}
extern "C" hipError_t algoCuda_main()
{
    return hipGetLastError();
}
/* OpenCV data format

    int limit = 10,
            channel = 3;
    unsigned char *input = img.data;
    for(int i = 0;i < img.rows && limit ;i++){
        for(int j = 0;j < img.cols * channel && limit; j += channel){
           uchar b = input[img.step * i + j] ;
           uchar g = input[img.step * i + j + 1];
           uchar r = input[img.step * i + j + 2];
            qDebug () << b << g << r;
            limit--;
        }
    }

    qDebug () << "next";
    for (int i = 0; i < 10; i++){
        cv::Vec3b intensity = img.at<cv::Vec3b>(0, i);
        uchar b = intensity.val[0];
        uchar g = intensity.val[1];
        uchar r = intensity.val[2];
        qDebug () << b << g << r;
    }

/*
 __global__ void kernCuda(float *Source,float * Destination, int width)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int i = abs(x*cosf(theta)-y*sinf(theta));
    int j = abs(x*sinf(theta)+y*cosf(theta));

    if(x<width && y<width){
        Destination[j*width+i]=Source[y*width+x];
    }

}
 */
