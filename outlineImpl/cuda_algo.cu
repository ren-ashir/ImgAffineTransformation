#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <ctime>
#include <cmath>

#define PI 3.14159265359

using namespace std;
//
// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}
// add
__global__ void sumKernel (float *a,float *b, float *c)
{
    //Global index of thread
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = a[i] + b[i];
}
template<class T>
class CudaArray
{
public:
    CudaArray (int n) : dev{nullptr},nBytes{0}
    {
        try{
            nBytes = n * sizeof(T);
            hipMalloc((void**)&dev,nBytes);
        } catch (...){}
    }
    ~CudaArray ()
    {
        try{
            hipFree (dev);
        }
        catch(...) {}
    }
    int getnBytes() const
    {
        return nBytes;
    }
    operator T* () const
    {
        return dev;
    }
    CudaArray& operator = (T* rhs) const
    {
        hipMemcpy (dev,rhs,nBytes,hipMemcpyHostToDevice);
    }
    void copyToCpu(T *cpu) const
    {
        hipMemcpy (cpu,dev,nBytes,hipMemcpyDeviceToHost);
    }
private:
    T *dev;
    int nBytes;
};
hipError_t sum (float *a,float *b,float*c,int n)
{
    CudaArray<float> aDev(n),bDev(n),cDev(n);
    dim3 threads = dim3(512,1);
    dim3 blocks = dim3(n / threads.x,1);
    aDev = a;
    bDev = b;
    sumKernel<<<blocks, threads>>> (aDev,bDev,cDev);
    cDev.copyToCpu(c);
    return hipGetLastError();
}
// tranform GPU
// add
__global__ void getTransformed_xy_Kernel (float *x,float *y)
{
    //Global index of thread
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    const float teta = PI / 2.5;
    const float sinteta = sin (teta);
    const float costeta = cos (teta);
    float xi = x[i];
    float yi = y[i];
    x[i] = xi * costeta + yi * sinteta;
    y[i] = -xi * sinteta + yi * costeta;
}
void  transform_xyData_gpu(float *x,float *y,int n)
{
    CudaArray<float> xDev(n),yDev(n);
    float *xn = new float [n],*yn = new float[n];
    dim3 threads = dim3(512,1);
    dim3 blocks = dim3(n / threads.x,1);
    xDev = x;
    yDev = y;
    getTransformed_xy_Kernel <<< blocks, threads >>> (xDev,yDev);
    xDev.copyToCpu(xn);
    yDev.copyToCpu(yn);
    delete [] xn;
    delete [] yn;
}
extern "C" hipError_t cuda_main()
{
    return hipGetLastError();
}

/*
 __global__ void kernCuda(float *Source,float * Destination, int width)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int i = abs(x*cosf(theta)-y*sinf(theta));
    int j = abs(x*sinf(theta)+y*cosf(theta));

    if(x<width && y<width){
        Destination[j*width+i]=Source[y*width+x];
    }

}
 */
